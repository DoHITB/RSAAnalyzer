#include "hip/hip_runtime.h"
/*
 * EQSolver.cu
 *
 *  Created on: 30 nov. 2020
 *      Author: DoHITB under MIT Liscense
 */
#include "conio.h"
#include "stdio.h"
#include "string.h"
#include "stdlib.h"
#include "time.h"

#include "hip/hip_runtime.h"
#include ""

/////////////////////////// FUNCTION DEFINITION ///////////////////////////
__device__ void CUnewBI(void* dst, char* s, int sig);
__device__ static void CUclean(void* va);
__device__ static int cu_strlen(char* s);
__device__ void CUiniStr(char** dst);
__device__ void CUtoString(void* vb, char* dst);
__device__ void CUsMul(void* va, void* vb, void* vc, void* vd);
__device__ void CUpAdd(void* va, void* vb);
__device__ void CUpSub(void* va, void* vb);
__device__ static void CUsubtract(void* va, void* vb);
__device__ static void CUcarrySub(void* va, int carryType);
__device__ static void CUrecount(void* va);
__device__ static void CUaddition(void* va, void* vb);
__device__ static void CUpMul(int pos, void* vpart);
__device__ void CUcarryAdd(void* va, int move, int min);
__device__ void CUhardEquals(void* va, void* vb, int* ret);
__device__ static int CUsignum(int a, int b);
__device__ void CUsub(void* va, void* vb, void* vc);
__device__ void CUsDvs(void* va, void* vb, void* xa, void* xb);
__device__ static void CUdivide(void* va, void* vb, void* xa, void* xb);
__device__ void CUpAppend(void* va, int b);
__device__ void CUsqrt(void* va, void* vb, void* vc, void* vd);

__global__ void kernelCal(void* x, void* y, void* n, void* a, void* b, void* c, void* d, void* e, void* f, void* xa, void* xb, void* num, int* arr, void* r);
__global__ void makeString(void* a, int count);
__device__ int checkSolution(void* va, void* vb, void* xa, void* xb, int* arr, void* num);
__global__ void propagateAdd(void* va, void* vb)
/////////////////////////// FUNCTION DEFINITION ///////////////////////////

/////////////////////////// BIGINTEGER MOCKUP ///////////////////////////
static void _BI_initialize();
void BImemcpy(void* dst, int value);
void newBI(void* dst, char* s, int sig);
static void clean(void* va);
void hardEquals(void* va, void* vb, int* ret);
static int signum(int a, int b);
void toString(void* vb, char* dst);

static int MAX_LENGTH = 1024;
//struct
struct BigInteger {
  char k;
  int count;
  char n[1024];
};

/*
 * Variables útiles
 */
static struct BigInteger _ZERO;
static struct BigInteger _ONE;
static struct BigInteger _TWO;
static struct BigInteger _THREE;
static struct BigInteger _FOUR;
static struct BigInteger _FIVE;
static struct BigInteger _SIX;
static struct BigInteger _SEVEN;
static struct BigInteger _EIGHT;
static struct BigInteger _NINE;
static struct BigInteger _TEN;
static struct BigInteger _HUND;
static struct BigInteger _MIN;
/////////////////////////// BIGINTEGER MOCKUP ///////////////////////////

int main(int argc, char* argv[]) {
  //////////// HOST DATA ////////////
  //file read
  FILE* fl;
  int efl;

  //kernel control
  int cn;
  int cm;

  if (argc > 1) {
    cn = atoi(argv[1]);
    cm = atoi(argv[2]);
  } else {
    cn = 1;
    cm = 1;
  }

  //heap memory control
  size_t maxMem;

  //loop control
  int ex = 0;

  //measurement
  float str;
  float end;
  float freq;

  //HOST data
  char* st1 = (char*)malloc(sizeof(char) * 1025);
  int* harr = (int*)malloc(sizeof(int) * 2);
  struct BigInteger* hnum = (struct BigInteger*)malloc(sizeof(struct BigInteger));
  struct BigInteger* ho = (struct BigInteger*)malloc(sizeof(struct BigInteger));
  struct BigInteger* ha = (struct BigInteger*)malloc(sizeof(struct BigInteger));
  struct BigInteger* hb = (struct BigInteger*)malloc(sizeof(struct BigInteger));
  struct BigInteger* hc = (struct BigInteger*)malloc(sizeof(struct BigInteger));
  struct BigInteger* hd = (struct BigInteger*)malloc(sizeof(struct BigInteger));
  struct BigInteger* he = (struct BigInteger*)malloc(sizeof(struct BigInteger));
  struct BigInteger* hf = (struct BigInteger*)malloc(sizeof(struct BigInteger));
  struct BigInteger* hn = (struct BigInteger*)malloc(sizeof(struct BigInteger) * cn * cm);
  struct BigInteger* zro = (struct BigInteger*)malloc(sizeof(struct BigInteger));
  struct BigInteger* hr = (struct BigInteger*)malloc(sizeof(struct BigInteger));
  int hi = 0;
  hipError_t cuerr;
  char* st = (char*)malloc(sizeof(char) * 1025);

  //////////// DEVICE DATA ////////////
  //BI Creation
  char* dst;
  struct BigInteger* a;
  struct BigInteger* b;
  struct BigInteger* c;
  struct BigInteger* d;
  struct BigInteger* e;
  struct BigInteger* f;
  struct BigInteger* n;
  struct BigInteger* o;
  struct BigInteger* x;
  struct BigInteger* y;
  struct BigInteger* num;
  int* arr;

  //AUX
  struct BigInteger* xa;
  struct BigInteger* xb;

  //Linkage
  struct BigInteger* r;

  _BI_initialize();
  newBI(zro, "0", 0);

  hipDeviceGetLimit(&maxMem, hipLimitMallocHeapSize);
  hipDeviceSetLimit(hipLimitMallocHeapSize, maxMem);
  printf("Heap memory set result: %s\n", hipGetErrorString(hipGetLastError()));
  printf("Kernel <<<%i, %i>>> will be launched\n", cn, cm);

  //get data from file
  efl = fopen_s(&fl, "cuda_var.dat", "r");

  if (efl != 0)
    exit(-1);

  //init DEVICE data
  hipMalloc(&dst, sizeof(char) * 1025);
  hipMalloc(&arr, sizeof(int) * 2);
  hipMalloc(&a, sizeof(struct BigInteger));
  hipMalloc(&b, sizeof(struct BigInteger));
  hipMalloc(&c, sizeof(struct BigInteger));
  hipMalloc(&d, sizeof(struct BigInteger));
  hipMalloc(&e, sizeof(struct BigInteger));
  hipMalloc(&f, sizeof(struct BigInteger));
  hipMalloc(&n, sizeof(struct BigInteger) * cn * cm);
  hipMalloc(&o, sizeof(struct BigInteger));
  hipMalloc(&x, sizeof(struct BigInteger) * cn * cm);
  hipMalloc(&y, sizeof(struct BigInteger) * cn * cm);
  hipMalloc(&num, sizeof(struct BigInteger));
  hipMalloc(&xa, sizeof(struct BigInteger) * cn * cm);
  hipMalloc(&xb, sizeof(struct BigInteger) * cn * cm);
  hipMalloc(&r, sizeof(struct BigInteger));

  printf("CLOCKS PER SECOND: %i\n", CLOCKS_PER_SEC);
  printf("Data allocation - Start @%i\n", clock());
  printf("\t Allocated memory:\n");
  printf("\t\t%i BigInteger\n", (5 * cn * cm) + 9);
  printf("\t\t2-position int array\n");
  printf("\t\t1025-position char array\n");

  //a, b, c, d, e, f
  fscanf_s(fl, "%s", st1, 1025);
  newBI(ha, st1, 0);

  fscanf_s(fl, "%s", st1, 1025);
  newBI(hb, st1, 0);

  fscanf_s(fl, "%s", st1, 1025);
  newBI(hc, st1, 0);

  fscanf_s(fl, "%s", st1, 1025);
  newBI(hd, st1, 0);

  fscanf_s(fl, "%s", st1, 1025);
  newBI(he, st1, 0);

  fscanf_s(fl, "%s", st1, 1025);
  newBI(hf, st1, 0);

  //arr
  fscanf_s(fl, "%i", &harr[0]);
  fscanf_s(fl, "%i", &harr[1]);

  //num
  fscanf_s(fl, "%s", st1, 5000);
  newBI(hnum, st1, 0);

  fclose(fl);

  //n (number)
  for (hi = 0; hi < (cn * cm); hi++) {
    _itoa_s(hi, st1, 10, 10);
    newBI(&hn[hi], st1, 0);
  }

  //o (offset)
  _itoa_s((cn * cm), st1, 10, 10);
  newBI(ho, st1, 0);

  //r (return)
  newBI(hr, "0", 0);

  //copy from HOTST to DEVICE
  hipMemcpy(a, ha, sizeof(struct BigInteger), hipMemcpyHostToDevice);           //1
  hipMemcpy(b, hb, sizeof(struct BigInteger), hipMemcpyHostToDevice);           //1
  hipMemcpy(c, hc, sizeof(struct BigInteger), hipMemcpyHostToDevice);           //1
  hipMemcpy(d, hd, sizeof(struct BigInteger), hipMemcpyHostToDevice);           //1
  hipMemcpy(e, he, sizeof(struct BigInteger), hipMemcpyHostToDevice);           //1
  hipMemcpy(f, hf, sizeof(struct BigInteger), hipMemcpyHostToDevice);           //1
  hipMemcpy(arr, harr, sizeof(int) * 2, hipMemcpyHostToDevice);                 //2 (int)
  hipMemcpy(num, hnum, sizeof(struct BigInteger), hipMemcpyHostToDevice);       //1
  hipMemcpy(n, hn, sizeof(struct BigInteger) * cn * cm, hipMemcpyHostToDevice); //cn * cm
  hipMemcpy(o, ho, sizeof(struct BigInteger), hipMemcpyHostToDevice);           //1
  hipMemcpy(r, hr, sizeof(struct BigInteger), hipMemcpyHostToDevice);           //1
  hipDeviceSynchronize();
  printf("Data allocation - End @%i\n", clock());

  //once reached here, the HOST memory can be free'd
  free(ha);
  free(hb);
  free(hc);
  free(hd);
  free(he);
  free(hf);
  free(ho);
  free(hn);
  free(harr);
  free(hnum);

  //init loop
  ex = 0;

  //calculation begins
  while (ex == 0) {
    printf("Calculation - Loop Start @%i\n", clock());
    str = clock();

    kernelCal <<<cn, cn>>>(x, y, n, a, b, c, d, e, f, xa, xb, num, arr, r);
    cuerr = hipDeviceSynchronize();

    if (cuerr != 0) {
      printf("Error on sync: %s\n", hipGetErrorString(cuerr));
      exit(-1);
    }

    end = clock();
    freq =  (cn * cm) / (end - str);
    printf("Calculation - Loop End @%i {%i values checked} ~ %f equations / msec\n", clock(), cn * cm, freq);

    //makeString<<<1, 1>>>(y, cn * cm);
    //copy r to HOST
    hipMemcpy(hr, r, sizeof(struct BigInteger), hipMemcpyDeviceToHost);
    hardEquals(hr, zro, &ex);

    propagateAdd <<<cn, cm>>>(n, o);
  }

  //Loop exit. Display result
  toString(hr, st1);
  printf("RESULT: %s\n", st1);

  return 0;
}


/////////////////////////// CALCULATION DEFINITION ///////////////////////////
//                        multi    multi    multi    single   single   single   single   single   single   multi     multi     single     2         single
__global__ void kernelCal(void* x, void* y, void* n, void* a, void* b, void* c, void* d, void* e, void* f, void* xa, void* xb, void* num, int* arr, void* r) {
  //x = d
  memcpy(&((struct BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x], d, sizeof(struct BigInteger));

  //x *= n
  CUsMul(&((struct BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

  //x += e
  CUpAdd(&((struct BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x], e);

  //y = n
  memcpy(&((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x],
         sizeof(struct BigInteger));

  //y *= n
  CUsMul(&((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

  //y *= c
  CUsMul(&((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x], c,
         &((struct BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

  //x += y
  CUpAdd(&((struct BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x]);

  //sqrt(x) > using y as temp var (will be overwritten)
  CUsqrt(&((struct BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

  //y = b
  memcpy(&((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x], b, sizeof(struct BigInteger));

  //y *= n
  CUsMul(&((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

  //y -= a
  CUsub(&((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x], a,
        &((struct BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x]);

  //y += x
  CUpAdd(&((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x]);

  //y /= f
  CUsDvs(&((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x], f,
         &((struct BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
         &((struct BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

  if (checkSolution(&((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
                    &((struct BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x],
                    &((struct BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
                    &((struct BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x],
                    arr, num) == 0)
    memcpy(r, &((struct BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x], sizeof(struct BigInteger));
}

//                         multi
__global__ void makeString(void* a, int count) { 
  int i = 0;
  char* st;
  CUiniStr(&st);

  for (i = 0; i < count; i++) {
    CUtoString(&((struct BigInteger*)a)[i], st);
    printf("[%i]: %s\n", i, st);
  }

  free(st);
}

//                           single    single    single    single    single    single
__device__ int checkSolution(void* va, void* vb, void* xa, void* xb, int* arr, void* num) {
  int y;

  //restamos 1 a va
  --((struct BigInteger*)va)->n[0];
  CUcarrySub(va, 0);
  CUrecount(va);

  //Desplazamos va una posición
  y = ((struct BigInteger*)va)->count + 1;

  for (; y >= 1; y--)
    ((struct BigInteger*)va)->n[y] = ((struct BigInteger*)va)->n[y - 1];

  ++((struct BigInteger*)va)->count;

  //copiamos num para tener el valor original
  memcpy(xa, num, sizeof(struct BigInteger));

  //probamos con arr[0]
  ((struct BigInteger*)va)->n[0] = arr[1];

  //dividimos como podemos (va sigue con la modificación)
  CUdivide(xa, va, vb, xb);

  //multiplicamos xa * va (va sigue con la modificación)
  CUsMul(xa, va, vb, xb);

  //comparamos xa contra num
  CUhardEquals(xa, num, &y);

  //si hay un resultado satisfactorio retornamos
  if (y == 0)
    return 0;

  //sino, probamos con arr[1]
  //copiamos num para tener el valor original
  memcpy(xa, num, sizeof(struct BigInteger));

  //dividimos como podemos (va sigue con la modificación)
  CUdivide(xa, va, vb, xb);

  //multiplicamos xa * va (va sigue con la modificación)
  CUsMul(xa, va, vb, xb);

  //comparamos xa contra num
  CUhardEquals(xa, num, &y);

  //retornamos y sea cual sea el resultado
  return y;
}

//                           multi     single
__global__ void propagateAdd(void* va, void* vb) {
  //va += vb
  CUpAdd(&((struct BigInteger*)va)[blockIdx.x * blockDim.x + threadIdx.x], vb);
}

/////////////////////////// MOCK-UP DEFINITION ///////////////////////////
__device__ void CUnewBI(void* dst, char* s, int sig) {
  int i = cu_strlen(s) - 1;
  int f = i;
  int j = 0;
  int c;
  int ssig = sig;

  //ajustamos el tipo
  ((struct BigInteger*)dst)->k = 'i';

  //limpiamos el array
  CUclean(dst);

  //recorremos el string y lo guardamos en integers
  for (; i >= 0; i--) {
    c = (int)(s[i] - 48);

    if (c >= 0 && c <= 9)
      ((struct BigInteger*)dst)->n[j++] = c;
    else
      if (s[i] == '-')
        ssig = -1;
  }

  //si nos envían un negativo, restamos una posición
  if (s[0] == '-')
    --f;

  ((struct BigInteger*)dst)->count = f;

  if (ssig == -1)
    ((struct BigInteger*)dst)->n[((struct BigInteger*)dst)->count] *= -1;
}

//mock-up for clean
__device__ static void CUclean(void* va) {
  int i = 0;

  for (i = 0; i < 1024; i++)
    ((struct BigInteger*)va)->n[i] = 0;

  ((struct BigInteger*)va)->count = 0;
}

//mock-up for iniStr
__device__ void CUiniStr(char** dst) {
  *dst = (char*)malloc(sizeof(char) * 1025);
}

//mock-up for toString
__device__ void CUtoString(void* vb, char* dst) {
  int i = 0;
  int m = ((struct BigInteger*)vb)->count;
  int sig = 0;

  //si el primer dígito es negativo, lo volvemos positivo y marcamos el caracter
  if (((struct BigInteger*)vb)->n[m] < 0) {
    ((struct BigInteger*)vb)->n[m] *= -1;
    dst[i++] = '-';
    sig = 1;
  }

  for (; m >= 0; m--)
    dst[i++] = (char)(((struct BigInteger*)vb)->n[m] + 48);

  dst[i] = '\0';

  //estamos trabajando sobre el puntero, revisamos y reestablecemos valores
  if (sig == 1)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
}

//mock-up for sMul
__device__ void CUsMul(void* va, void* vb, void* vc, void* vd) {
  int sig;
  int i;
  int x;

  CUclean(vc);
  CUclean(vd);

  sig = CUsignum(((struct BigInteger*)va)->n[((struct BigInteger*)va)->count],
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count]);

  if (sig == 1)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  else if (sig == 10)
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;
  else if (sig == 11) {
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  }

  for (i = 0; i <= ((struct BigInteger*)vb)->count; i++) {
    for (x = 0; x <= ((struct BigInteger*)va)->count; x++)
      ((struct BigInteger*)vc)->n[x] = ((struct BigInteger*)va)->n[x] * ((struct BigInteger*)vb)->n[i];

    ((struct BigInteger*)vc)->count = x - 1;
    CUcarryAdd(vc, 0, 0);

    //multiplicación parcial
    CUpMul(i, vc);

    //sumamos
    CUaddition(vd, vc);
  }

  //copiamos resultado
  memcpy(va, vd, sizeof(struct BigInteger));

  //si los signos son diferentes, invertimos el signo
  if (sig == 1 || sig == 10)
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;

  //desnormalizamos vb
  if (sig == 1)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  else if (sig == 11)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;

  CUrecount(va);
}

//mock-up for pAdd
__device__ void CUpAdd(void* va, void* vb) {
  int sig = CUsignum(((struct BigInteger*)va)->n[((struct BigInteger*)va)->count],
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count]);

  //normalizamos los operandos
  if (sig == 10)
    //a negativo, b positivo. Cambiamos el signo de "a" y hacemos suma
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;
  else if (sig == 1)
    //b negativo, a positivo. Cambiamos el signo de "b" y hacemos suma
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  else if (sig == 11) {
    //a negativo, b negativo. Cambiamos signos y hacemos suma
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  }

  //si ambos signos son iguales, se suma, sino, se resta
  if (sig == 0 || sig == 11)
    CUaddition(va, vb);
  else
    CUsubtract(va, vb);

  if (sig == 10 || sig == 11)
    //en estos casos, siempre se le va la vuelta al signo
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;

  //desnormalizamos b
  if (sig == 1)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  else if (sig == 11)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
}

//mock-up for pSub
__device__ void CUpSub(void* va, void* vb) {
  int sig;

  sig = CUsignum(((struct BigInteger*)va)->n[((struct BigInteger*)va)->count],
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count]);

  //normalizamos los signos
  if (sig == 1)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  else if (sig == 10)
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;
  else if (sig == 11) {
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  }

  //si tienen el mismo signo, se resta, sino se suma
  if (sig == 0 || sig == 11)
    CUsubtract(va, vb);
  else
    CUaddition(va, vb);

  if (sig == 10 || sig == 11)
    //en estos casos, siempre se le va la vuelta al signo
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;

  //desnormalizamos b
  if (sig == 1)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  else if (sig == 11)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
}

//mock-up for subtract
__device__ static void CUsubtract(void* va, void* vb) {
  int i = 0;

  //restamos los dígitos comunes
  for (; i <= ((struct BigInteger*)vb)->count; i++)
    ((struct BigInteger*)va)->n[i] -= ((struct BigInteger*)vb)->n[i];

  //si el último dígito es negativo
  if (((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] < 0)
    CUcarrySub(va, 1);
  else
    CUcarrySub(va, 0); 
}

//mock-up for carrySub
__device__ static void CUcarrySub(void* va, int carryType) {
  int i = 0;
  int acc = 0;

  if (carryType == 0) {
    for (; i <= ((struct BigInteger*)va)->count; i++) {
      //restamos el acarreo al número
      ((struct BigInteger*)va)->n[i] -= acc;

      if (((struct BigInteger*)va)->n[i] < 0) {
        //normalizamos el número
        ((struct BigInteger*)va)->n[i] += 10;
        acc = 1;
      } else
        acc = 0;
    }
  } else {
    //en esta opción, no es necesario pasar una segunda vez por acarreos.
    for (i = 0; i < ((struct BigInteger*)va)->count; i++)
      if (((struct BigInteger*)va)->n[i] < 0)
        //normalizamos el número
        ((struct BigInteger*)va)->n[i] = ((struct BigInteger*)va)->n[i] * -1;
  }

  //contamos de nuevo los dígitos
  CUrecount(va);
}

//mock-up for recount
__device__ static void CUrecount(void* va) {
  while (((struct BigInteger*)va)->n[((struct BigInteger*)va)->count--] == 0);

  ++((struct BigInteger*)va)->count;

  if (((struct BigInteger*)va)->count < 0)
    ((struct BigInteger*)va)->count = 0;
}

//mock-up for addition
__device__ static void CUaddition(void* va, void* vb) {
  int limit;
  int min;
  int swap;
  int move;
  int i;

  //asumimos que a tiene la mayor longitud
  limit = ((struct BigInteger*)va)->count;

  //asumimos que b tiene la menor longitud
  min = ((struct BigInteger*)vb)->count;

  //indicador de necesidad de arrastre
  move = 0;
  i = 0;

  //si no es así, rectificamos
  if (((struct BigInteger*)vb)->count > limit) {
    //intercambiamos limit y min
    swap = limit;
    limit = min;
    min = swap;

    move = 1;
  }

  //sumamos los dígitos que coinciden
  for (; i <= min; i++)
    ((struct BigInteger*)va)->n[i] += ((struct BigInteger*)vb)->n[i];

  //los dígitos que no coinciden los traspasamos
  if (move == 1) {
    for (; i <= limit; i++)
      ((struct BigInteger*)va)->n[i] = ((struct BigInteger*)vb)->n[i];

    ((struct BigInteger*)va)->count = limit;
  }

  //gestionamos el acarreo
  CUcarryAdd(va, 1, min);
}

//mock-up for pMul
__device__ static void CUpMul(int pos, void* vpart) {
  int i;

  i = ((struct BigInteger*)vpart)->count + pos;

  //generamos offset
  for (; i >= pos; i--)
    ((struct BigInteger*)vpart)->n[i] = ((struct BigInteger*)vpart)->n[i - pos];

  //noramlizamos las posiciones de offset
  for (i = 0; i < pos; i++)
    ((struct BigInteger*)vpart)->n[i] = 0;

  ((struct BigInteger*)vpart)->count += pos;
}

//mock-up for carryAdd
__device__ void CUcarryAdd(void* va, int move, int min) {
  int i = 0;
  int acc;
  int limit;

  acc = 0;

  //move == 1 --> sabemos que hay una parte no común. En min está el límite de la parte común
  if (move == 1)
    limit = min;
  else
    limit = ((struct BigInteger*)va)->count;

  //recorremos a y vamos sumando el acarreo de la parte común
  for (; i <= limit; i++) {
    //sumamos acarreo
    ((struct BigInteger*)va)->n[i] += acc;

    //como acc es int, podemos dividir entre 10 y sacar el acarreo
    acc = ((struct BigInteger*)va)->n[i] / 10;

    if (acc > 0)
      //normalizamos el número
      ((struct BigInteger*)va)->n[i] = ((struct BigInteger*)va)->n[i] % 10;
  }

  if (move == 1) {
    //queda parte no común. Acarreamos hatsa que acc sea 0, ya que la parte no común ya está normalizada
    while (acc > 0 && i <= ((struct BigInteger*)va)->count) {
      //sumamos acarreo
      ((struct BigInteger*)va)->n[i] += acc;

      //como acc es int, podemos dividir entre 10 y sacar el acarreo
      acc = ((struct BigInteger*)va)->n[i] / 10;

      if (acc > 0)
        //normalizamos el número
        ((struct BigInteger*)va)->n[i] = ((struct BigInteger*)va)->n[i] % 10;

      i++;
    }
  }

  //si ha quedado acarreo, lo guardamos al final;
  if (acc > 0)
    ((struct BigInteger*)va)->n[++(((struct BigInteger*)va)->count)] = acc;
}

//mock-up for hardEquals
__device__ void CUhardEquals(void* va, void* vb, int* ret) {
  int i;
  int sig;

  /*calculamos el signo
   *
   *00: a >= 0, b >= 0
   *01: a >= 0, b <  0
   *10: a <  0, b >= 0
   *11: a <  0, b <  0
   */
  sig = CUsignum(((struct BigInteger*)va)->n[((struct BigInteger*)va)->count],
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count]);

  if (sig == 1)
    //a >= 0, b < 0, por tanto a > b
    *ret = 1;
  else if (sig == 10)
    //a < 0, b >= 0, por tanto a < b
    *ret = 2;
  else {
    //comparten signo. Hacemos comparación manual
    *ret = 0;

    if (((struct BigInteger*)va)->count < ((struct BigInteger*)vb)->count)
      //si "a" tiene menos dígitos que "b"
      *ret = 2;
    else if (((struct BigInteger*)va)->count > ((struct BigInteger*)vb)->count)
      //si "a" tiene más dítigos que "b"
      *ret = 1;
    else {
      //tienen los mismos dígitos. Comparamos uno a uno.
      for (i = ((struct BigInteger*)va)->count; i >= 0; i--) {
        if (((struct BigInteger*)va)->n[i] < ((struct BigInteger*)vb)->n[i])
          *ret = 2;
        else if ((((struct BigInteger*)va)->n[i] > ((struct BigInteger*)vb)->n[i]))
          *ret = 1;

        if (*ret > 0)
          break;
      }
    }

    if (sig == 11) {
      //ambos tienen signo negativo. Cambiamos el retorno
      //ya que se p. ej. -1 > -10
      if (*ret == 2)
        *ret = 1;
      else if (*ret == 1)
        *ret = 2;
    }
  }
}

//mock-up for signum
__device__ static int CUsignum(int a, int b) {
  int ret = 0;

  if (a < 0)
    ret = 10;

  if (b < 0)
    ++ret;

  return ret;
}

//mock-up for sDvs
__device__ void CUsDvs(void* va, void* vb, void* xa, void* xb) {
  int sig;
  int comp;

  CUnewBI(xa, "0", 0);
  CUnewBI(xb, "1", 0);

  sig = CUsignum(((struct BigInteger*)va)->n[((struct BigInteger*)va)->count],
               ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count]);

  //normalizamos los operandos
  if (sig == 1)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  else if (sig == 10)
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;
  else if (sig == 11) {
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  }

  CUhardEquals(va, vb, &comp);

  if (comp == 0) {
    //si a = b, a/b = 1
    CUhardEquals(va, xa, &comp);

    if (comp == 0)
      //si a = 0 (por tanto, b = 0), retornamos 0
      memcpy(va, xa, sizeof(struct BigInteger));
    else
      //en caso contrario, retornamos 1
      memcpy(va, xb, sizeof(struct BigInteger));
  } else if (comp == 2) {
    //si a < b, a / b = 0
    memcpy(va, xa, sizeof(struct BigInteger));
  } else if (comp == 1) {
    //si a > b, buscamos un número que solucione ax = n
    CUhardEquals(vb, xb, &comp);

    if (comp != 0)
      CUdivide(va, vb, xa, xb);
  }

  //si los signos son diferentes, invertimos el signo
  if (sig == 1 || sig == 10)
    ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] *= -1;

  //desnormalizamos b
  if (sig == 1)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
  else if (sig == 11)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
}

//mock-up for divide
__device__ static void CUdivide(void* va, void* vb, void* xa, void* xb) {
  int len;
  int i = 0;
  int x = 0;
  int res = 0;
  int mLen;
  int eq;

  len = ((struct BigInteger*)va)->count - ((struct BigInteger*)vb)->count;

  CUclean(xa);
  CUclean(xb);

  /*
   * Nos quedamos con los "b.count" primeros dígitos. Si "b" tiene un único
   * dígito, no movemos nada, porque ya moveremos el siguiente dígito más adelante
   */
  if (((struct BigInteger*)vb)->count > 0)
    for (; i < ((struct BigInteger*)vb)->count; i++)
      ((struct BigInteger*)xb)->n[((struct BigInteger*)vb)->count - i - 1] = ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count - i];
  else
    ((struct BigInteger*)xb)->n[0] = 0;

  ((struct BigInteger*)xb)->count = ((struct BigInteger*)vb)->count - 1;

  if (((struct BigInteger*)xb)->count == -1)
    ++((struct BigInteger*)xb)->count;

  //si "b" tiene una cifra, b.len será 0 pero tenemos que restar una cifra igualmente
  if (((struct BigInteger*)vb)->count == 0)
    mLen = len - 1;
  else
    mLen = len;

  //por cada cifra decimal que hemos generado
  for (i = 0; i <= len; i++) {
    CUpAppend(xb, ((struct BigInteger*)va)->n[len - i]);

    for (x = 0; x < 10; x++) {
      if (((struct BigInteger*)xb)->n[((struct BigInteger*)xb)->count] == 0) {
        //el valor es 0
        res = x;
        x = 99;
      } else if (((struct BigInteger*)xb)->n[((struct BigInteger*)xb)->count] < 0) {
        //el valor es negativo
        CUaddition(xb, vb);
        res = (x - 1);
        x = 99;
      } else {
        CUhardEquals(xb, vb, &eq);

        if (eq == 2) {
          //xb < vb --> found
          res = x;
          x = 99;
        } else {
          //xb >= vb
          CUsubtract(xb, vb);
        }
      }
    }

    ((struct BigInteger*)xa)->n[len - i] = res;
  }

  ((struct BigInteger*)xa)->count = mLen;

  memcpy(va, (struct BigInteger*)xa, sizeof(struct BigInteger));

  CUrecount(va);

  //evitar error de falso desbordamiento
  if (((struct BigInteger*)va)->n[((struct BigInteger*)va)->count + 1] > 0)
    ++((struct BigInteger*)va)->count;
}

//mock-up for pAppend
__device__ void CUpAppend(void* va, int b) {
  int i;

  for (i = ((struct BigInteger*)va)->count; i >= 0; i--)
    ((struct BigInteger*)va)->n[i + 1] = ((struct BigInteger*)va)->n[i];

  ((struct BigInteger*)va)->n[0] = b;
  ++((struct BigInteger*)va)->count;
}

//mock-up for sub
__device__ void CUsub(void* va, void* vb, void* vc) {
  int comp;
  int sig;

  CUhardEquals(va, vb, &comp);
  sig = CUsignum(((struct BigInteger*)va)->n[((struct BigInteger*)va)->count],
               ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count]);

  //si ambos son negativos, comp = 1 significa a < b
  if ((comp == 2 && sig < 11) || (comp == 1 && sig == 11)) {
    memcpy(vc, vb, sizeof(struct BigInteger)); //tmp = b; stmp = m[0]

    CUpSub(vc, va);

    //cambiamos el signo
    //((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
    ((struct BigInteger*)vc)->n[((struct BigInteger*)vc)->count] *= -1;

    //reasignamos
    memcpy(va, vc, sizeof(struct BigInteger));

    //rescatamos el valor original de vb
    //memcpy(vb, &((struct BigInteger*)m)[0], sizeof(struct BigInteger));
  } else {
    CUpSub(va, vb);
  }
}

__device__ void CUsqrt(void* va, void* vb, void* vc, void* vd) {
  char even = ((struct BigInteger*)va)->count % 2;
  int digits = ((struct BigInteger*)va)->count + 1;
  char fstep;
  int x = 0;
  int y = 0;
  int eq;
  char z;

  CUclean(vb);
  CUclean(vc);
  CUclean(vd);

  ///// PRIMER RESTO /////
  //creamos el primer resto
  if (even == 1) {
    //si even == 1 es que count es impar y por tanto hay un numero par de cifras
    fstep = ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count] * 10 + ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count - 1];
  } else {
    //sino, es un numero impar de cifras
    fstep = ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count];
  }

  //calculamos la raíz simple
  while (x * x++ <= fstep);
  if (--x * x > fstep)
    --x;

  //x es el primer dígito de la raiz
  ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] = x;

  //guardaremos el resto en vc. Siempre se dará fstep >= x
  ((struct BigInteger*)vc)->n[((struct BigInteger*)vc)->count] = fstep - (x * x);

  //recontamos
  CUrecount(vc);
  ///// PRIMER RESTO /////

  //utilizamos "digits" como contador. 
  if (digits == 1 || digits == 2) {
    //caso especial, para 1 y 2 dígitos el resultado tiene 1 dígito
    digits = 0;
  } else {
    //el count incluye el 0, restamos 1
    digits = (digits / 2 + digits % 2) - 1;
  }

  //usamos "x" como contador temporal, sumando 1 porque ya hemos colocado 1 dígito
  x = 1;

  ///// RESTO DE DÍGITOS /////
  //mientras haya dígitos por colocar
  while (x <= digits) {
    //bajamos las dos siguientes cifras.
    //  primero corremos dos posiciones el resto (vc)
    y = ((struct BigInteger*)vc)->count + 2;

    for (; y >= 2; y--)
      ((struct BigInteger*)vc)->n[y] = ((struct BigInteger*)vc)->n[y - 2];

    //  luego movemos de va a vc las dos posiciones correlativas
    ((struct BigInteger*)vc)->n[1] = ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count - (2 * x) - even + 1];
    ((struct BigInteger*)vc)->n[0] = ((struct BigInteger*)va)->n[((struct BigInteger*)va)->count - (2 * x) - even];

    // ajustamos la longitud de vc
    ((struct BigInteger*)vc)->count += 2;

    //ajustamos
    CUrecount(vc);

    //copiamos el resultado actual (vb) a vd
    memcpy(vd, vb, sizeof(struct BigInteger));

    //calculamos vb + vb. Da igual el órden en que sumemos
    for (y = 0; y <= ((struct BigInteger*)vd)->count; y++)
      ((struct BigInteger*)vd)->n[y] += ((struct BigInteger*)vd)->n[y];

    //gesionamos acarreo
    CUcarryAdd(vd, 0, 0);

    //buscamos un número n = [1, 9] tal que vd * 10 + n * n = vc
    // primero montamos el valor relativo a n = 1
    y = ((struct BigInteger*)vd)->count + 1;

    for (; y >= 1; y--)
      ((struct BigInteger*)vd)->n[y] = ((struct BigInteger*)vd)->n[y - 1];

    ((struct BigInteger*)vd)->n[0] = 1;
    ++((struct BigInteger*)vd)->count;

    //inicializamos z, que será la cifra a colocar
    z = 1;

    //asumimos que no llegamos a igualar vb
    eq = 2;

    ///// BUSQUEDA DEL DÍGITO /////
    while (eq == 2) {
      //comparmos vd con vb
      CUhardEquals(vd, vc, &eq);

      if (eq == 1) {
        //vd > vb. La cifra que buscamos es la anterior.
        //restauramos vd al estado anterior
        if (z == 1) {
          //si z = 1 estamos en la primera iteración, y el valor anterior será 0
          CUnewBI(vd, "0", 0);
          --z;
        } else {
          //restamos z y restauramos vd. El decremento de vd es (vd * 10) + (2(n - 1) + 1)
          --z;
        }
      } else if (eq == 2) {
        //vd < vb. La cifra que buscamos puede ser la siguiente
        //incrementamos vd y z. El incremento de vd es (vd * 10) + (2n + 1).
        //Obtenemos la primera parte de vb

        //(vd * 10)
        for (y = 0; y <= ((struct BigInteger*)vb)->count; y++)
          ((struct BigInteger*)vd)->n[y + 1] += (((struct BigInteger*)vb)->n[y] * 2);

        //+2n + 1
        ((struct BigInteger*)vd)->n[0] += (2 * z) + 1;

        //gestionamos acarreo
        CUcarryAdd(vd, 0, 0);

        ++z;
      }

      //si hemos iterado 10 veces, la cifra será 9
      if (z == 10) {
        //volvemos a 9
        --z;

        //forzamos salida
        eq = 1;
      }
    }
    ///// BUSQUEDA DEL DÍGITO /////

    ///// RESTAURAR /////
    if (eq == 1) {
      //si nos hemos pasado, restauramos el valor de vd
      //copiamos el resultado actual (vb) a vd
      memcpy(vd, vb, sizeof(struct BigInteger));

      //calculamos vb + vb. Da igual el órden en que sumemos
      for (y = 0; y <= ((struct BigInteger*)vd)->count; y++)
        ((struct BigInteger*)vd)->n[y] += ((struct BigInteger*)vd)->n[y];

      //gesionamos acarreo
      CUcarryAdd(vd, 0, 0);

      //buscamos un número n = [1, 9] tal que vd * 10 + n * n = vc
      // primero montamos el valor relativo a n = 1
      y = ((struct BigInteger*)vd)->count + 1;

      for (; y >= 1; y--)
        ((struct BigInteger*)vd)->n[y] = ((struct BigInteger*)vd)->n[y - 1];

      ((struct BigInteger*)vd)->n[0] = z;
      ++((struct BigInteger*)vd)->count;

      //multiplicamos por z
      for (y = 0; y <= ((struct BigInteger*)vd)->count; y++)
        ((struct BigInteger*)vd)->n[y] *= z;

      //gestionamos acarreo
      CUcarryAdd(vd, 0, 0);
    }

    ///// COLOCAR DÍTIGTO /////
    //Al salir del bucle tendremos en "z" la cifra a colocar, en vd el dato a restar, y en vc el cociente actual
    //Restamos vc -= vd (siempre se dará vc >= vd) para actualizar el cociente
    CUsubtract(vc, vd);

    //Desplazamos vb una posición y colocamos z
    y = ((struct BigInteger*)vb)->count + 1;

    for (; y >= 1; y--)
      ((struct BigInteger*)vb)->n[y] = ((struct BigInteger*)vb)->n[y - 1];

    ((struct BigInteger*)vb)->n[0] = z;
    ++((struct BigInteger*)vb)->count;
    ///// COLOCAR DÍTIGTO /////

    //incrementamos x
    x++;
  }
  ///// RESTO DE DÍGITOS /////

  //copiamos el resultado
  memcpy(va, vb, sizeof(struct BigInteger));
}
/////////////////////////// MOCK-UP DEFINITION ///////////////////////////

/////////////////////////// AUX DEFINITION ///////////////////////////
__device__ static int cu_strlen(char* s) {
  int ret = 0;

  while (*s != '\0') {
    ++s;
    ++ret;
  }

  return ret;
}
/////////////////////////// AUX DEFINITION ///////////////////////////

/////////////////////////// BIGINTEGER DEFINITION ///////////////////////////
static void _BI_initialize() {
  int i = 0;

  //lo creamos manualmente ya que clean copia de _ZERO
  for (; i < MAX_LENGTH; i++)
    _ZERO.n[i] = 0;

  _ZERO.count = 0;
  _ZERO.k = 'i';

  newBI(&_ONE, "1", 0);
  newBI(&_TWO, "2", 0);
  newBI(&_THREE, "3", 0);
  newBI(&_FOUR, "4", 0);
  newBI(&_FIVE, "5", 0);
  newBI(&_SIX, "6", 0);
  newBI(&_SEVEN, "7", 0);
  newBI(&_EIGHT, "8", 0);
  newBI(&_NINE, "9", 0);
  newBI(&_TEN, "10", 0);
  newBI(&_HUND, "100", 0);
  newBI(&_MIN, "1", -1);
}

void BImemcpy(void* dst, int value) {
  if (value == 0)
    memcpy(dst, &_ZERO, sizeof(struct BigInteger));
  else if (value == 1)
    memcpy(dst, &_ONE, sizeof(struct BigInteger));
  else if (value == 2)
    memcpy(dst, &_TWO, sizeof(struct BigInteger));
  else if (value == 3)
    memcpy(dst, &_THREE, sizeof(struct BigInteger));
  else if (value == 4)
    memcpy(dst, &_FOUR, sizeof(struct BigInteger));
  else if (value == 5)
    memcpy(dst, &_FIVE, sizeof(struct BigInteger));
  else if (value == 6)
    memcpy(dst, &_SIX, sizeof(struct BigInteger));
  else if (value == 7)
    memcpy(dst, &_SEVEN, sizeof(struct BigInteger));
  else if (value == 8)
    memcpy(dst, &_EIGHT, sizeof(struct BigInteger));
  else if (value == 9)
    memcpy(dst, &_NINE, sizeof(struct BigInteger));
  else if (value == 10)
    memcpy(dst, &_TEN, sizeof(struct BigInteger));
  else if (value == 100)
    memcpy(dst, &_HUND, sizeof(struct BigInteger));
  else if (value == -1)
    memcpy(dst, &_MIN, sizeof(struct BigInteger));
  else
    BImemcpy(dst, 0);
}

void newBI(void* dst, char* s, int sig) {
  int i = (int)strlen(s) - 1;
  int f = i;
  int j = 0;
  int c;
  int ssig = sig;

  //ajustamos el tipo
  ((struct BigInteger*)dst)->k = 'i';

  //limpiamos el array
  clean(dst);

  //recorremos el string y lo guardamos en integers
  for (; i >= 0; i--) {
    c = (int)(s[i] - 48);

    if (c >= 0 && c <= 9) {
      ((struct BigInteger*)dst)->n[j++] = c;
    } else {
      if (s[i] == '-')
        ssig = -1;
    }
  }

  //si nos envían un negativo, restamos una posición
  if (s[0] == '-')
    --f;

  ((struct BigInteger*)dst)->count = f;

  if (ssig == -1)
    ((struct BigInteger*)dst)->n[((struct BigInteger*)dst)->count] *= -1;
}

static void clean(void* va) {
  BImemcpy(va, 0);
}

void hardEquals(void* va, void* vb, int* ret) {
  int i;
  int sig;

  /*calculamos el signo
   *
   *00: a >= 0, b >= 0
   *01: a >= 0, b <  0
   *10: a <  0, b >= 0
   *11: a <  0, b <  0
   */
  sig = signum(((struct BigInteger*)va)->n[((struct BigInteger*)va)->count],
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count]);

  if (sig == 1)
    //a >= 0, b < 0, por tanto a > b
    *ret = 1;
  else if (sig == 10)
    //a < 0, b >= 0, por tanto a < b
    *ret = 2;
  else {
    //comparten signo. Hacemos comparación manual
    *ret = 0;

    if (((struct BigInteger*)va)->count < ((struct BigInteger*)vb)->count)
      //si "a" tiene menos dígitos que "b"
      *ret = 2;
    else if (((struct BigInteger*)va)->count > ((struct BigInteger*)vb)->count)
      //si "a" tiene más dítigos que "b"
      *ret = 1;
    else {
      //tienen los mismos dígitos. Comparamos uno a uno.
      for (i = ((struct BigInteger*)va)->count; i >= 0; i--) {
        if (((struct BigInteger*)va)->n[i] < ((struct BigInteger*)vb)->n[i])
          *ret = 2;
        else if ((((struct BigInteger*)va)->n[i] > ((struct BigInteger*)vb)->n[i]))
          *ret = 1;

        if (*ret > 0)
          break;
      }
    }

    if (sig == 11) {
      //ambos tienen signo negativo. Cambiamos el retorno
      //ya que se p. ej. -1 > -10
      if (*ret == 2)
        *ret = 1;
      else if (*ret == 1)
        *ret = 2;
    }
  }
}

static int signum(int a, int b) {
  int ret = 0;

  if (a < 0)
    ret = 10;

  if (b < 0)
    ++ret;

  return ret;
}

void toString(void* vb, char* dst) {
  int i = 0;
  int m = ((struct BigInteger*)vb)->count;
  int sig = 0;

  //si el primer dígito es negativo, lo volvemos positivo y marcamos el caracter
  if (((struct BigInteger*)vb)->n[m] < 0) {
    ((struct BigInteger*)vb)->n[m] *= -1;
    dst[i++] = '-';
    sig = 1;
  }

  for (; m >= 0; m--)
    dst[i++] = (char)(((struct BigInteger*)vb)->n[m] + 48);

  dst[i] = '\0';

  //estamos trabajando sobre el puntero, revisamos y reestablecemos valores
  if (sig == 1)
    ((struct BigInteger*)vb)->n[((struct BigInteger*)vb)->count] *= -1;
}
/////////////////////////// BIGINTEGER DEFINITION ///////////////////////////
