#include "hip/hip_runtime.h"
/*
 * EQSolver.cu
 *
 *  Created on: 30 nov. 2020
 *      Author: DoHITB under MIT Liscense
 * 
 *  CHANGELOG
 *    v1.1: General revision. Several bugs found, code identation, comments, etc.
 *          Added a small improvement on performance by checking remainder of sqrt.
 *          Use of BigInteger.cu (custom!)
 *          Added mcd usage
 *          Added hyperbolic axis limit
 *          Added memory check
 */
#include "conio.h"
#include "stdio.h"
#include "string.h"
#include "stdlib.h"
#include "time.h"
#include "BigInteger.h"

#include "hip/hip_runtime.h"
#include ""

 /////////////////////////// FUNCTION DEFINITION ///////////////////////////
void mcd(void* a, void* b, void* m);
__global__ void kernelCal(void* x, void* y, void* n, void* a, void* b, void* c, void* d, void* e, void* f, void* xa, void* xb, void* o, void* m, void* ax, void* num, int* arr, volatile void* r);
__device__ int checkSolution(void* va, void* vb, void* xa, void* xb, int* arr, void* num);

int main(int argc, char* argv[]) {
  //////////// HOST DATA ////////////
  //file read
  FILE* fl;
  int efl;

  //kernel control
  int cn;
  int cm;

  if (argc > 1) {
    cn = atoi(argv[1]);
    cm = atoi(argv[2]);
  } else {
    cn = 32;
    cm = 4096;
  }

  //heap memory control
  size_t maxMem;

  //HOST data
  char* st1 = (char*)malloc(sizeof(char) * MAX_LENGTH);
  int* harr = (int*)malloc(sizeof(int) * 2);
  BigInteger* hnum = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* ho = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* ha = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* hb = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* hc = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* hd = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* he = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* hf = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* hn = (BigInteger*)malloc(sizeof(BigInteger) * cn * cm);
  BigInteger* zro = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* hr = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* hmcd = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* hmce = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* haxis = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* hvalue = (BigInteger*)malloc(sizeof(BigInteger));
  int hi = 0;
  hipError_t cuerr;
  char* st = (char*)malloc(sizeof(char) * MAX_LENGTH);
  memory* hm = (memory*)malloc(getMemorySize());

  //////////// DEVICE DATA ////////////
  //BI Creation
  char* dst;
  BigInteger* a;
  BigInteger* b;
  BigInteger* c;
  BigInteger* d;
  BigInteger* e;
  BigInteger* f;
  BigInteger* n;
  BigInteger* o;
  BigInteger* x;
  BigInteger* y;
  BigInteger* m;
  BigInteger* ax;
  BigInteger* vl;
  BigInteger* num;
  int* arr;

  //AUX
  BigInteger* xa;
  BigInteger* xb;

  //Linkage
  BigInteger* r;

  init((void**)hm);
  newBI(zro, "0", 0);

  hipDeviceGetLimit(&maxMem, hipLimitMallocHeapSize);
  hipDeviceSetLimit(hipLimitMallocHeapSize, maxMem);
  printf("Heap memory set result: %s\n", hipGetErrorString(hipGetLastError()));
  printf("Kernel <<<%i, %i>>> will be launched\n", cn, cm);

  //get data from file
  efl = fopen_s(&fl, "cuda_var.dat", "r");

  if (efl != 0)
    exit(-1);

  //init DEVICE data
  cuerr = hipMalloc(&dst, sizeof(char) * MAX_LENGTH);

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&arr, sizeof(int) * 2);

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&a, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&b, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&c, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&d, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&e, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&f, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&n, sizeof(BigInteger) * cn * cm);

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&o, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&x, sizeof(BigInteger) * cn * cm);

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&y, sizeof(BigInteger) * cn * cm);

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&num, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&xa, sizeof(BigInteger) * cn * cm);

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&xb, sizeof(BigInteger) * cn * cm);

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&r, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&m, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&ax, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMalloc(&vl, sizeof(BigInteger));

  if (cuerr != 0) {
    printf("Error on malloc: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  printf("CLOCKS PER SECOND: %i\n", CLOCKS_PER_SEC);
  printf("Data allocation - Start @%i\n", clock());
  printf("\t Allocated memory:\n");
  printf("\t\t%i BigInteger\n", (5 * cn * cm) + 12);
  printf("\t\t2-position int array\n");
  printf("\t\t1025-position char array\n");

  //a, b, c, d, e, f
  fscanf_s(fl, "%s", st1, MAX_LENGTH);
  newBI(ha, st1, 0);

  fscanf_s(fl, "%s", st1, MAX_LENGTH);
  newBI(hb, st1, 0);

  fscanf_s(fl, "%s", st1, MAX_LENGTH);
  newBI(hc, st1, 0);

  fscanf_s(fl, "%s", st1, MAX_LENGTH);
  newBI(hd, st1, 0);

  fscanf_s(fl, "%s", st1, MAX_LENGTH);
  newBI(he, st1, 0);

  fscanf_s(fl, "%s", st1, MAX_LENGTH);
  newBI(hf, st1, 0);

  //arr
  fscanf_s(fl, "%i", &harr[0]);
  fscanf_s(fl, "%i", &harr[1]);

  //num
  fscanf_s(fl, "%s", st1, MAX_LENGTH);
  newBI(hnum, st1, 0);

  fscanf_s(fl, "%s", st1, MAX_LENGTH);
  newBI(haxis, st1, 0);

  fscanf_s(fl, "%s", st1, MAX_LENGTH);
  newBI(hvalue, st1, 0);

  fclose(fl);

  //n (number)
  for (hi = 0; hi < (cn * cm); hi++) {
    _itoa_s(hi, st1, 10, 10);
    newBI(&hn[hi], st1, 0);
  }

  //o (offset)
  _itoa_s((cn * cm), st1, 10, 10);
  newBI(ho, st1, 0);

  //r (return)
  newBI(hr, "0", 0);

  //Before copying to device, reduce the number by using mcd
  memcpy(hmcd, he, sizeof(BigInteger));                                    //hmcd = he;
  memcpy(hmce, hd, sizeof(BigInteger));                                    //hmce = hd;
  mcd(hmcd, hmce, hm);                                                     //hmcd = mcd(hmcd, hmce);

  memcpy(hmce, hc, sizeof(BigInteger));                                    //hmce = hc;
  mcd(hmcd, hmce, hm);                                                     //hmcd = mcd(hmcd, hmce);

  //Once we have mcd, divide all three values by mcd
  dvs(he, hmcd, hm);                                                       //he /= hmcd;
  dvs(hd, hmcd, hm);                                                       //hd /= hmcd;
  dvs(hc, hmcd, hm);                                                       //hc /= hmcd;

  //copy from HOTST to DEVICE
  cuerr = hipMemcpy(a, ha, sizeof(BigInteger), hipMemcpyHostToDevice);           //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(b, hb, sizeof(BigInteger), hipMemcpyHostToDevice);           //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(c, hc, sizeof(BigInteger), hipMemcpyHostToDevice);           //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(d, hd, sizeof(BigInteger), hipMemcpyHostToDevice);           //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(e, he, sizeof(BigInteger), hipMemcpyHostToDevice);           //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(f, hf, sizeof(BigInteger), hipMemcpyHostToDevice);           //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(arr, harr, sizeof(int) * 2, hipMemcpyHostToDevice);          //2 (int)

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(num, hnum, sizeof(BigInteger), hipMemcpyHostToDevice);       //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(n, hn, sizeof(BigInteger) * cn * cm, hipMemcpyHostToDevice); //cn * cm

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(o, ho, sizeof(BigInteger), hipMemcpyHostToDevice);           //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(r, hr, sizeof(BigInteger), hipMemcpyHostToDevice);           //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(m, hmcd, sizeof(BigInteger), hipMemcpyHostToDevice);         //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(ax, haxis, sizeof(BigInteger), hipMemcpyHostToDevice);         //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipMemcpy(vl, hvalue, sizeof(BigInteger), hipMemcpyHostToDevice);         //1

  if (cuerr != 0) {
    printf("Error on copy: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  cuerr = hipDeviceSynchronize();
  printf("Data allocation - End @%i\n", clock());

  //once reached here, the HOST memory can be free'd
  free(ha);
  free(hb);
  free(hc);
  free(hd);
  free(he);
  free(hf);
  free(ho);
  free(hn);
  free(harr);
  free(hnum);
  free(hmcd);
  free(hmce);
  free(haxis);
  free(hvalue);

  kernelCal <<<cn, cn >>>(x, y, n, a, b, c, d, e, f, xa, xb, o, m, ax, num, arr, r);
  cuerr = hipDeviceSynchronize();

  if (cuerr != 0) {
    printf("Error on sync: %s\n", hipGetErrorString(cuerr));
    exit(-1);
  }

  //copy r to HOST
  hipMemcpy(hr, r, sizeof(BigInteger), hipMemcpyDeviceToHost);

  //check if found
  equals(hr, zro, &hi);
  toString(hr, st1);

  if (hi == 0) {
    printf("Result don't found under hiperbolic limit\n");
    printf("Factor is between [%s, 1]\n", st1);
  } else {
    printf("RESULT: %s\n", st1);
  }
  
  return 0;
}


void mcd(void* a, void* b, void* m) {
  BigInteger* t = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* z = (BigInteger*)malloc(sizeof(BigInteger));
  BigInteger* d = (BigInteger*)malloc(sizeof(BigInteger));

  int* r = (int*)malloc(sizeof(int));

  BImemcpy(z, 0);                           //z = 0;

  equals(a, b, r);                          //a > b?

  if (*r == 2) {
    //a < b, change order
    memcpy(d, a, sizeof(BigInteger));       //d = a;
    memcpy(a, b, sizeof(BigInteger));       //a = b;
    memcpy(b, d, sizeof(BigInteger));       //b = d;
  }

  equals(b, z, r);                          //b = 0?

  while (*r != 0) {
    memcpy(t, b, sizeof(BigInteger));       //  t = b;
    mod(a, b, m);                           //  a %= b;
    memcpy(b, a, sizeof(BigInteger));       //  b = a;
    memcpy(a, t, sizeof(BigInteger));       //  a = t;
    equals(b, z, r);                        //  b = 0?
  }

  free(t);
  free(z);
  free(d);
  free(r);
}

/////////////////////////// CALCULATION DEFINITION ///////////////////////////
//                        multi    multi    multi    single   single   single   single   single   single   multi     multi     single   single   single     single     2         single
__global__ void kernelCal(void* x, void* y, void* n, void* a, void* b, void* c, void* d, void* e, void* f, void* xa, void* xb, void* o, void* m, void* ax, void* num, int* arr, volatile void* r) {
  int ret;

  //check for axis value
  CUhardEquals(&((BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x], ax, &ret);

  //while(r == 0 || ret == 2) {ret = 2 means n < axis}
  while ((((BigInteger*)r)->count == 0 && ((BigInteger*)r)->n[0] == 0) || (ret == 2)) {
    //x = d
    memcpy(&((BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x], d, sizeof(BigInteger));

    //x *= n
    CUsMul(&((BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

    //x += e
    CUpAdd(&((BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x], e);

    //y = n
    memcpy(&((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x],
      sizeof(BigInteger));

    //y *= n
    CUsMul(&((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

    //y *= c
    CUsMul(&((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x], c,
      &((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

    //x += y
    CUpAdd(&((BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x]);

    //x *= m
    CUsMul(&((BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x],
      m,
      &((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

    //sqrt(x) > using y as temp var (will be overwritten)
    CUsqrt(&((BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
      &((BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

    //hack-way-check remainder of sqrt(x). Only continue if "x" is a perfect square (remainder = 0)
    if (((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x].count == 0 &&
      ((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x].n[0] == 0) {
      //y = b
      memcpy(&((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x], b, sizeof(BigInteger));

      //y *= n
      CUsMul(&((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
        &((BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x],
        &((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
        &((BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

      //y -= a
      CUsub(&((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x], a,
        &((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x]);

      //y += x
      CUpAdd(&((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
        &((BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x]);

      //y /= f
      CUsDvs(&((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x], f,
        &((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
        &((BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x]);

      if (checkSolution(&((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x],
        &((BigInteger*)x)[blockIdx.x * blockDim.x + threadIdx.x],
        &((BigInteger*)xa)[blockIdx.x * blockDim.x + threadIdx.x],
        &((BigInteger*)xb)[blockIdx.x * blockDim.x + threadIdx.x],
        arr, num) == 0)
        memcpy((void*)r, &((BigInteger*)y)[blockIdx.x * blockDim.x + threadIdx.x], sizeof(BigInteger));
    }

    //check for axis value
    CUhardEquals(&((BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x], ax, &ret);

    //n += o
    CUpAdd(&((BigInteger*)n)[blockIdx.x * blockDim.x + threadIdx.x], o);
  }
}

//                           single    single    single    single    single    single
__device__ int checkSolution(void* va, void* vb, void* xa, void* xb, int* arr, void* num) {
  int y;
  int i = 0;
  int acc = 0;

  //subtract 1 from va
  --((BigInteger*)va)->n[0];
  //CUcarrySub(va, 0);
  //CUrecount(va);

  //Mockup for carrysub and recount as compiler is failing to include these functions????
  for (i = 0; i <= ((BigInteger*)va)->count; i++) {
    //subtract carry to number
    ((BigInteger*)va)->n[i] -= acc;

    if (((BigInteger*)va)->n[i] < 0) {
      //normalize
      ((BigInteger*)va)->n[i] += 10;
      acc = 1;
    } else
      acc = 0;
  }

  while (((BigInteger*)va)->n[((BigInteger*)va)->count--] == 0);

  ++((BigInteger*)va)->count;

  if (((BigInteger*)va)->count < 0)
    ((BigInteger*)va)->count = 0;

  //Offset by 1 position
  y = ((BigInteger*)va)->count + 1;

  for (; y >= 1; y--)
    ((BigInteger*)va)->n[y] = ((BigInteger*)va)->n[y - 1];

  ++((BigInteger*)va)->count;

  //copy num to get original value
  memcpy(xa, num, sizeof(BigInteger));

  //try with arr[0]
  ((BigInteger*)va)->n[0] = arr[1];

  //divide it (va still has modification)
  CUsDvs(xa, va, vb, xb);

  //compute xa * va (va still modified)
  CUsMul(xa, va, vb, xb);

  //compare xa vs num
  CUhardEquals(xa, num, &y);

  //if they're equal, return
  if (y == 0)
    return 0;

  //else, try with arr[1]
  //copy num to get original value
  memcpy(xa, num, sizeof(BigInteger));

  //divide it (va still has modification)
  CUsDvs(xa, va, vb, xb);

  //compute xa * va (va still modified)
  CUsMul(xa, va, vb, xb);

  //compare xa vs num
  CUhardEquals(xa, num, &y);

  //return y whatever the result is
  return y;
}
